#include "hip/hip_runtime.h"
#include "lab2.h"


static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info)
{
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

__global__ void simple_kernel(uint8_t *pos, unsigned int width, unsigned int height, float time)

{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	//x = x / (float) width;
	//y = y / (float) height;
	float v = x*6.0f - 1.0f;
	float u = y*2.5f - 1.0f;


    // calculate simple sine wave pattern

	float freq = 4.345f;
	float w = sinf(x*freq + time) * cosf(y*freq + time) * 4.5f;

	// write output vertex

    if(time<120)
    pos[y*width + x] = (uint8_t)(u*v+time)*987 % 256;
    if(time>120)
    pos[y*width + x] = (uint8_t)(w+u)*987 % 256;
}

void Lab2VideoGenerator::Generate(uint8_t*yuv)
{

	dim3 block(1, 1, 1);
	dim3 grid(W / block.x, H*1.5 / block.y, 1);
	simple_kernel <<< grid, block >>>(yuv, W, H,(impl->t));

	//int brightness1;
	//int brightness2;
	//brightness1=(rand()%255)+1;
	//brightness2=(rand()%255)+1;

	//int color;
	//color=(rand()%128)+1;

	//hipMemset(yuv,          (impl->t)*brightness1/NFRAME, W*H/2);
	//hipMemset(yuv+W*H/2,    (impl->t)*brightness2/NFRAME, W*H/2);
	//hipMemset(yuv+W*H,      color, W*H/2);

	++(impl->t);
}

