#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "SyncedMemory.h"
#include ""

using namespace std;



__global__ void SomeTransform(char *input_gpu, int fsize) 

{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize && input_gpu[idx] != '\n') 
	{
		input_gpu[idx] = input_gpu[idx]+input_gpu[idx];//copy two same part
	}
}


int main(int argc, char **argv)
{

	// init, and check
	if (argc != 2) 
	{
		printf("Usage %s <input text file>\n", argv[0]);
		
	}
	
	FILE *fp = fopen(argv[1], "r");
	if (not fp) 
	{
		printf("Cannot open %s", argv[1]);
		
	}
	
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize + 1);
	auto text_smem = text.CreateSync(fsize);

	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();


	SomeTransform<<<2, 32>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	
	printf("%d" ,text_smem );
	
	
	return 0;

}









dim3 block(1, 1, 1);

dim3 grid(W / block.x, H / block.y, 1);

simple_kernel << < grid, block >> >(yuv, W, H, impl->t);


__global__ void simple_kernel

(uint8_t *pos, unsigned int width, unsigned int height, float time)

{

unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

pos[y*width + x] = (uint8_t)(x) *(int)time % 256;

}


































