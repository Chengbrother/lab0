#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone
(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f)
	{
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb)
		{
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed//算出fixed
(
const float *background,const float *target,const float *mask,float *fixed,
const int wb, const int hb, const int wt, const int ht,const int oy, const int ox
)

{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	const int yb = oy+yt, xb = ox+xt;
	const int curb = wb*yb+xb;

	float nb0=0;
	float wbe0=0;
	float sb0=0;
	float eb0=0;

	float nb1=0;
	float wbe1=0;
	float sb1=0;
	float eb1=0;

	float nb2=0;
	float wbe2=0;
	float sb2=0;
	float eb2=0;


	float nt0=0;
	float wte0=0;
	float st0=0;
	float et0=0;

	float nt1=0;
	float wte1=0;
	float st1=0;
	float et1=0;

	float nt2=0;
	float wte2=0;
	float st2=0;
	float et2=0;


	if (mask[curt] < 127.0f)
		{
		}
	else
		{
			//n
			if(curt>=wt)
			{
				if(mask[(curt-wt)] > 127.0f)
				{
				}
				else
				{
					nb0=background[(curb-wb)*3+0];
					nb1=background[(curb-wb)*3+1];
					nb2=background[(curb-wb)*3+2];
				}
					nt0=target[(curt-wt)*3+0];
					nt1=target[(curt-wt)*3+1];
					nt2=target[(curt-wt)*3+2];
			 }
				else
				{
					nt0=target[(curt)*3+0];
					nt1=target[(curt)*3+1];
					nt2=target[(curt)*3+2];
					nb0=background[(curb-wb)*3+0];
					nb1=background[(curb-wb)*3+1];
					nb2=background[(curb-wb)*3+2];
				}

			//w
			if (curt%wt != 0)
			 {
                if(mask[(curt-1)] > 127.0f)
                {
                }
                else
                {
                	wbe0=background[(curb-1)*3+0];
                	wbe1=background[(curb-1)*3+1];
                	wbe2=background[(curb-1)*3+2];
                }
                	wte0=target[(curt-1)*3+0];
                	wte1=target[(curt-1)*3+1];
                	wte2=target[(curt-1)*3+2];
             }
                else
                {
                	wte0=target[curt*3+0];
                	wte1=target[curt*3+1];
                	wte2=target[curt*3+2];
                	wbe0=background[(curb-1)*3+0];
                	wbe1=background[(curb-1)*3+1];
                	wbe2=background[(curb-1)*3+2];
                }

            //s
            if (curt + wt<wt*ht)
              {
                if(mask[(curt+wt)] > 127.0f)
                {
                }
                else
                {
                	sb0=background[(curb+wb)*3+0];
                	sb1=background[(curb+wb)*3+1];
                	sb2=background[(curb+wb)*3+2];
                }
                	st0=target[(curt+wt)*3+0];
                	st1=target[(curt+wt)*3+1];
                	st2=target[(curt+wt)*3+2];
              	}
                else
                {
                	st0=target[curt*3+0];
                	st1=target[curt*3+1];
                	st2=target[curt*3+2];
                	sb0=background[(curb+wb)*3+0];
                	sb1=background[(curb+wb)*3+1];
                	sb2=background[(curb+wb)*3+2];

                }

             //e
             if ((curt + 1) % wt != 0)
              {
                if(mask[(curt+1)] > 127.0f)
                {
                }
                else
                {
                	eb0=background[(curb+1)*3+0];
                	eb1=background[(curb+1)*3+1];
                	eb2=background[(curb+1)*3+2];
                }
                	et0=target[(curt+1)*3+0];
                	et1=target[(curt+1)*3+1];
                	et2=target[(curt+1)*3+2];
              }
                else
                {
                	et0=target[curt*3+0];
                	et1=target[curt*3+1];
                	et2=target[curt*3+2];
                	eb0=background[(curb+1)*3+0];
                	eb1=background[(curb+1)*3+1];
                	eb2=background[(curb+1)*3+2];
                }
                fixed[curt*3+0]=4*target[curt*3+0]-nt0-st0-wte0-et0+nb0+sb0+wbe0+eb0;
                fixed[curt*3+1]=4*target[curt*3+1]-nt1-st1-wte1-et1+nb1+sb1+wbe1+eb1;
                fixed[curt*3+2]=4*target[curt*3+2]-nt2-st2-wte2-et2+nb2+sb2+wbe2+eb2;
			}
		}





__global__ void PoissonImageCloningIteration
(
const float *fixed, const float *mask, float *bufnow, float *bufnext, const int wt, const int ht
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	int curt = wt*yt+xt;

	float nbw0=0;
	float wbw0=0;
	float sbw0=0;
	float ebw0=0;

	float nbw1=0;
	float wbw1=0;
	float sbw1=0;
	float ebw1=0;

	float nbw2=0;
	float wbw2=0;
	float sbw2=0;
	float ebw2=0;
					//n
					if(curt>=wt)
					{
						if(mask[(curt-wt)] > 127.0f)
						{
							nbw0=bufnow[(curt-wt)*3+0];
							nbw1=bufnow[(curt-wt)*3+1];
							nbw2=bufnow[(curt-wt)*3+2];
						}
					}

					//w
					if (curt%wt !=0)
					{
		                if(mask[(curt-1)] > 127.0f)
		                {
		                	wbw0=bufnow[(curt-1)*3+0];
		                	wbw1=bufnow[(curt-1)*3+1];
		                	wbw2=bufnow[(curt-1)*3+2];
		                }
					}

					//s
		           if (curt+wt<wt*ht)
		           	   {

		                if(mask[(curt+wt)] > 127.0f)
		                {
		                	sbw0=bufnow[(curt+wt)*3+0];
		                	sbw1=bufnow[(curt+wt)*3+1];
		                	sbw2=bufnow[(curt+wt)*3+2];
		                }

		           	   	}

		           //e
		           if ((curt+1)%wt!=0)
		           {
		                if(mask[(curt+1)] > 127.0f)
		                {
		                	ebw0=bufnow[(curt+1)*3+0];
		                	ebw1=bufnow[(curt+1)*3+1];
		                	ebw2=bufnow[(curt+1)*3+2];
		                }
		            }
		           	   	bufnext[curt*3+0]= (fixed[curt*3+0]+ (nbw0+wbw0+sbw0+ebw0))/4;
		                bufnext[curt*3+1]= (fixed[curt*3+1]+ (nbw1+wbw1+sbw1+ebw1))/4;
		                bufnext[curt*3+2]= (fixed[curt*3+2]+ (nbw2+wbw2+sbw2+ebw2))/4;

			}
void PoissonImageCloning
(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	//set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));

	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);

	CalculateFixed<<<gdim, bdim>>>
			(
			background, target, mask, fixed,
			wb, hb, wt, ht, oy, ox
			);

	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

	// iterate

	for(int i=0;i<3000;++i)
	{

		dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);

		PoissonImageCloningIteration<<<gdim, bdim>>>(
				fixed, mask, buf1, buf2, wt, ht
				);


	    PoissonImageCloningIteration<<<gdim, bdim>>>(
				fixed, mask, buf2, buf1, wt, ht
			   );
	}


	//copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);



	SimpleClone<<<gdim, bdim>>>
	(
	   background, buf1, mask, output,
	   wb, hb, wt, ht, oy, ox
	);

	//SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		//background, target, mask, output,
		//wb, hb, wt, ht, oy, ox
	//);


	//clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}

